#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <hip/hip_runtime.h>

/*
Macro for checking CUDA code
LLM recommended it. Might move to utils if it comes in handy.
*/
#ifndef CUDA_CHECK
#define CUDA_CHECK(expr)                                                       \
	do {                                                                       \
		hipError_t _err = (expr);                                             \
		if (_err != hipSuccess) {                                             \
			throw std::runtime_error(                                          \
			    std::string("CUDA error: ") + hipGetErrorString(_err) +       \
			    " @ " + __FILE__ + ":" + std::to_string(__LINE__));            \
		}                                                                      \
	} while (0)
#endif

int cuda_hamming_distance(const std::string& seq1, const std::string& seq2) {
    const int len1 = seq1.length();
    const int len2 = seq2.length();
    const int maxLen = std::max(len1, len2);

    // Pad sequences with '-'
    std::string padded1 = seq1 + std::string(maxLen - len1, '-');
    std::string padded2 = seq2 + std::string(maxLen - len2, '-');

	// Pointers of sequences
	char *d_seq1 = nullptr, *d_seq2 = nullptr;
    int *d_result;
    int result = 0;

	// Allocate GPU mem
	CUDA_CHECK(hipMalloc((void **)&d_seq1, max_len * sizeof(char)));
	CUDA_CHECK(hipMalloc((void **)&d_seq2, max_len * sizeof(char)));
	CUDA_CHECK(hipMalloc((void **)&d_res, sizeof(int)));

	// Copy seq1 and seq2 and init H to 0
	CUDA_CHECK(hipMemcpy(d_seq1, padded1.data(), max_len * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_seq2, padded2.data(), max_len * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_seq2, &result, sizeof(int), hipMemcpyHostToDevice));

    // Copy result back
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(d_result);

    return result;

}

// int main() {
//     std::string seq1 = "aTGACsd";
//     std::string seq2 = "ATGAC";

//     std::cout << "Hamming distance: " << cuda_hamming_distance(seq1, seq2) << std::endl;

//     return 0;
// }
